#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>

#include <hermes/hermes.h>

using namespace hermes::cuda;

///////////////////////////////////////////////////////////////////////////////
////////////////////       INTERPOLATION     //////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

TEST_CASE("Interpolation", "[numeric][interpolation]") {
  SECTION("monotonicCubic") {
    { // 1D test
      auto f = [](float x) -> float { return cos(x) * sin(x); };
      for (float s = 0.0; s <= 1.0; s += 0.01) {
        REQUIRE(monotonicCubicInterpolate(f(-0.1), f(0.0), f(0.1), f(0.2), s) ==
                Approx(f(s * 0.1)).margin(0.1 * 0.1));
      }
      for (float s = 0.0; s <= 1.0; s += 0.01)
        REQUIRE(
            monotonicCubicInterpolate(f(-0.01), f(0.0), f(0.01), f(0.02), s) ==
            Approx(f(s * 0.01)).margin(0.01 * 0.01));
    }
  }
}

///////////////////////////////////////////////////////////////////////////////
////////////////////       GRID              //////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

struct map_ipj {
  __device__ void operator()(index2 index, float &value) const {
    value = index.i * 10 + index.j;
  }
};

TEST_CASE("Grid", "[numeric][grid][access]") {
  SECTION("2d") {
    Grid2<float> g(size2(10, 10), vec2(0.1, 0.1), point2(1, 2));
    g = 3.0;
    ponos::Grid2<float> hg = g.hostData();
    for (auto e : hg.accessor())
      REQUIRE(e.value == Approx(3).margin(1e-8));
    g.map(map_ipj());
    hg = g.hostData();
    for (auto e : hg.accessor())
      REQUIRE(e.value == Approx(e.index.i * 10 + e.index.j).margin(1e-8));
    Grid2<float> gg;
    gg = g;
    REQUIRE(g.resolution() == gg.resolution());
    REQUIRE(g.spacing() == gg.spacing());
    REQUIRE(g.origin() == gg.origin());
    g = 7.3f;
    hg = g.hostData();
    for (auto e : hg.accessor())
      REQUIRE(e.value == Approx(7.3).margin(1e-8));
    Array2<float> a(size2(10, 10));
    a = 1.f;
    g = a;
    hg = g.hostData();
    for (auto e : hg.accessor())
      REQUIRE(e.value == Approx(1).margin(1e-8));
  }
}

TEST_CASE("VectorGrid", "[numeric][grid]") {
  SECTION("2d") {
    SECTION("constructors operatiors") {
      VectorGrid2<float> empty;
      REQUIRE(empty.resolution() == size2(0, 0));
      empty = VectorGrid2<float>(size2(10, 10), vec2f(1));
      REQUIRE(empty.resolution() == size2(10, 10));
      VectorGrid2<float> g = empty;
      REQUIRE(g.resolution() == size2(10, 10));
      auto g2 = std::move(VectorGrid2<float>(size2(7, 7), vec2f(1)));
      REQUIRE(g2.resolution() == size2(7, 7));
      std::vector<VectorGrid2<float>> gs;
      gs.emplace_back(size2(10, 10), vec2(1));
      gs.emplace_back(size2(7, 7), vec2(1));
      std::vector<VectorGrid2<float>> gs2 = gs;
    }
    SECTION("CELL CENTERED") {
      VectorGrid2<float> vg(size2(10, 10), vec2(1));
      REQUIRE(vg.resolution() == size2(10, 10));
      REQUIRE(vg.u().resolution() == size2(10, 10));
      REQUIRE(vg.v().resolution() == size2(10, 10));
      REQUIRE(vg.origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.spacing().x == Approx(1).margin(1e-8));
      REQUIRE(vg.spacing().y == Approx(1).margin(1e-8));
      REQUIRE(vg.u().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.u().origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().y == Approx(0).margin(1e-8));
      VectorGrid2<float> vg2;
      vg2 = vg;
      REQUIRE(vg2.resolution() == size2(10, 10));
      REQUIRE(vg2.u().resolution() == size2(10, 10));
      REQUIRE(vg2.v().resolution() == size2(10, 10));
      REQUIRE(vg2.origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg2.origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg2.spacing().x == Approx(1).margin(1e-8));
      REQUIRE(vg2.spacing().y == Approx(1).margin(1e-8));
      REQUIRE(vg2.u().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg2.u().origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg2.v().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg2.v().origin().y == Approx(0).margin(1e-8));
      vg.setGridType(ponos::VectorGridType::STAGGERED);
      REQUIRE(vg.resolution() == size2(10, 10));
      REQUIRE(vg.u().resolution() == size2(11, 10));
      REQUIRE(vg.v().resolution() == size2(10, 11));
      REQUIRE(vg.origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.spacing().x == Approx(1).margin(1e-8));
      REQUIRE(vg.spacing().y == Approx(1).margin(1e-8));
      REQUIRE(vg.u().origin().x == Approx(-0.5).margin(1e-8));
      REQUIRE(vg.u().origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().y == Approx(-0.5).margin(1e-8));
    }
    SECTION("STAGGERED") {
      VectorGrid2<float> vg(ponos::VectorGridType::STAGGERED);
      vg.setResolution(size2(10, 10));
      REQUIRE(vg.resolution() == size2(10, 10));
      REQUIRE(vg.u().resolution() == size2(11, 10));
      REQUIRE(vg.v().resolution() == size2(10, 11));
      REQUIRE(vg.origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.spacing().x == Approx(1).margin(1e-8));
      REQUIRE(vg.spacing().y == Approx(1).margin(1e-8));
      REQUIRE(vg.u().origin().x == Approx(-0.5).margin(1e-8));
      REQUIRE(vg.u().origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().y == Approx(-0.5).margin(1e-8));
      VectorGrid2<float> vg2;
      vg2 = vg;
      REQUIRE(vg2.resolution() == size2(10, 10));
      REQUIRE(vg2.u().resolution() == size2(11, 10));
      REQUIRE(vg2.v().resolution() == size2(10, 11));
      REQUIRE(vg2.origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg2.origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg2.spacing().x == Approx(1).margin(1e-8));
      REQUIRE(vg2.spacing().y == Approx(1).margin(1e-8));
      REQUIRE(vg2.u().origin().x == Approx(-0.5).margin(1e-8));
      REQUIRE(vg2.u().origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg2.v().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg2.v().origin().y == Approx(-0.5).margin(1e-8));
      vg.setGridType(ponos::VectorGridType::CELL_CENTERED);
      REQUIRE(vg.resolution() == size2(10, 10));
      REQUIRE(vg.u().resolution() == size2(10, 10));
      REQUIRE(vg.v().resolution() == size2(10, 10));
      REQUIRE(vg.origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.spacing().x == Approx(1).margin(1e-8));
      REQUIRE(vg.spacing().y == Approx(1).margin(1e-8));
      REQUIRE(vg.u().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.u().origin().y == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().x == Approx(0).margin(1e-8));
      REQUIRE(vg.v().origin().y == Approx(0).margin(1e-8));
    }
  }
}

struct map_sincos {
  map_sincos(Grid2Accessor<float> acc) : acc(acc) {}
  __device__ void operator()(index2 index, float &value) const {
    auto wp = acc.worldPosition(index);
    value = sin(wp.x) * cos(wp.y);
  }
  Grid2Accessor<float> acc;
};

TEST_CASE("VectorGridAccessor", "[numeric][grid][accessor]") {
  auto f = [](ponos::point2 wp) -> float { return sin(wp.x) * cos(wp.y); };
  SECTION("2d") {
    SECTION("methods") {
      ponos::VectorGrid2<float> hg(ponos::size2(10), ponos::vec2(0.1),
                                   ponos::point2(),
                                   ponos::VectorGridType::STAGGERED);
      VectorGrid2<float> dg(size2(10), vec2(0.1), point2(),
                            ponos::VectorGridType::STAGGERED);
      auto hacc = hg.accessor();
      auto dacc = dg.accessor();
      for (index2 ij : Index2Range<i32>(dg.resolution())) {
        REQUIRE(hacc.u().worldPosition(ij.ponos()).x ==
                Approx(dacc.u().worldPosition(ij).x).margin(1e-8));
        REQUIRE(hacc.u().worldPosition(ij.ponos()).y ==
                Approx(dacc.u().worldPosition(ij).y).margin(1e-8));
        REQUIRE(hacc.v().worldPosition(ij.ponos()).x ==
                Approx(dacc.v().worldPosition(ij).x).margin(1e-8));
        REQUIRE(hacc.v().worldPosition(ij.ponos()).y ==
                Approx(dacc.v().worldPosition(ij).y).margin(1e-8));
      }
    }
    SECTION("device to host") {
      VectorGrid2<float> dg(size2(10), vec2f(0.01));
      dg.u().map(map_sincos(dg.u().accessor()));
      dg.v().map(map_sincos(dg.v().accessor()));
      auto hg = dg.hostData();
      auto dacc = dg.accessor();
      auto hacc = hg.accessor();
      for (index2 ij : Index2Range<i32>(dg.resolution())) {
        auto v = hacc[ij.ponos()];
        REQUIRE(v.x == Approx(f(hacc.worldPosition(ij.ponos()))));
        REQUIRE(v.y == Approx(f(hacc.worldPosition(ij.ponos()))));
        REQUIRE(hacc.u().worldPosition(ij.ponos()).x ==
                Approx(dacc.u().worldPosition(ij).x).margin(1e-8));
        REQUIRE(hacc.u().worldPosition(ij.ponos()).y ==
                Approx(dacc.u().worldPosition(ij).y).margin(1e-8));
        REQUIRE(hacc.v().worldPosition(ij.ponos()).x ==
                Approx(dacc.v().worldPosition(ij).x).margin(1e-8));
        REQUIRE(hacc.v().worldPosition(ij.ponos()).y ==
                Approx(dacc.v().worldPosition(ij).y).margin(1e-8));
      }
    }
    SECTION("host to device") {
      ponos::VectorGrid2<float> hg(ponos::size2(10), ponos::vec2(0.1),
                                   ponos::point2(),
                                   ponos::VectorGridType::STAGGERED);
      for (auto e : hg.u().accessor())
        e.value = f(e.worldPosition());
      for (auto e : hg.v().accessor())
        e.value = f(e.worldPosition());

      VectorGrid2<float> dg = hg;

      auto hdg = dg.hostData();
      auto hacc = hg.accessor();
      auto dacc = dg.accessor();
      for (index2 ij : Index2Range<i32>(dg.resolution())) {
        auto v = hacc[ij.ponos()];
        REQUIRE(v.x == Approx((f(hacc.u().worldPosition(ij.ponos())) +
                               f(hacc.u().worldPosition(ij.right().ponos()))) /
                              2));
        REQUIRE(v.y == Approx((f(hacc.v().worldPosition(ij.ponos())) +
                               f(hacc.v().worldPosition(ij.up().ponos()))) /
                              2));
        REQUIRE(hacc.u().worldPosition(ij.ponos()).x ==
                Approx(dacc.u().worldPosition(ij).x).margin(1e-8));
        REQUIRE(hacc.u().worldPosition(ij.ponos()).y ==
                Approx(dacc.u().worldPosition(ij).y).margin(1e-8));
        REQUIRE(hacc.v().worldPosition(ij.ponos()).x ==
                Approx(dacc.v().worldPosition(ij).x).margin(1e-8));
        REQUIRE(hacc.v().worldPosition(ij.ponos()).y ==
                Approx(dacc.v().worldPosition(ij).y).margin(1e-8));
      }
    }
  }
}