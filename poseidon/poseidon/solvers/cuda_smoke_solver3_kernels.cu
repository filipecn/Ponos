#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 FilipeCN
 *
 * The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * iM the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

#include <poseidon/math/cuda_pcg.h>
#include <poseidon/solvers/cuda_smoke_solver3_kernels.h>

namespace poseidon {

namespace cuda {

texture<float, hipTextureType3D> uTex3, uCopyTex3, wuTex3;
texture<float, hipTextureType3D> vTex3, vCopyTex3, wvTex3;
texture<float, hipTextureType3D> wTex3, wCopyTex3, wwTex3;
texture<float, hipTextureType3D> densityTex3;
texture<float, hipTextureType3D> pressureTex3;
texture<float, hipTextureType3D> divergenceTex3;
texture<unsigned char, hipTextureType3D> solidTex3;
texture<float, hipTextureType3D> uSolidTex3;
texture<float, hipTextureType3D> vSolidTex3;
texture<float, hipTextureType3D> wSolidTex3;
texture<float, hipTextureType3D> forceTex3;
texture<float, hipTextureType3D> temperatureTex3;

using namespace hermes::cuda;

__global__ void __injectTemperature(RegularGrid3Accessor<float> t,
                                    RegularGrid3Accessor<float> tTarget,
                                    float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (t.isIndexStored(x, y, z))
    t(x, y, z) += (1 - expf(-dt)) * (tTarget(x, y, z) - t(x, y, z));
}

template <>
void injectTemperature(
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, float> &temperature,
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, float>
        &targetTemperature,
    float dt) {
  hermes::ThreadArrayDistributionInfo td(temperature.resolution());
  __injectTemperature<<<td.gridSize, td.blockSize>>>(
      temperature.accessor(), targetTemperature.accessor(), dt);
}

__global__ void __injectSmoke(RegularGrid3Accessor<float> s,
                              RegularGrid3Accessor<unsigned char> source,
                              float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (s.isIndexStored(x, y, z)) {
    s(x, y, z) += dt * source(x, y, z);
    s(x, y, z) = fminf(1.f, s(x, y, z));
  }
}

template <>
void injectSmoke(
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, float> &smoke,
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, unsigned char> &source,
    float dt) {
  hermes::ThreadArrayDistributionInfo td(smoke.resolution());
  __injectSmoke<<<td.gridSize, td.blockSize>>>(smoke.accessor(),
                                               source.accessor(), dt);
}

__global__ void __applyForceField(RegularGrid3Accessor<float> velocity,
                                  RegularGrid3Accessor<float> force, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (velocity.isIndexStored(x, y, z)) {
    auto wp = velocity.worldPosition(x, y, z);
    auto p = force.gridPosition(wp) + vec3f(0.5f);
    // printf("%f ", tex3D(forceTex3, p.x, p.y, p.z));
    velocity(x, y, z) += dt * tex3D(forceTex3, p.x, p.y, p.z);
  }
}

template <>
void applyForceField(StaggeredGrid3D &velocity, VectorGrid3D &forceField,
                     float dt) {
  forceTex3.filterMode = hipFilterModeLinear;
  forceTex3.normalized = 0;
  Array3<float> forceArray(forceField.u().resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(forceTex3, forceArray.data(), channelDesc));
  {
    memcpy(forceArray, forceField.u().data());
    hermes::ThreadArrayDistributionInfo td(velocity.u().resolution());
    __applyForceField<<<td.gridSize, td.blockSize>>>(
        velocity.u().accessor(), forceField.u().accessor(), dt);
  }
  {
    memcpy(forceArray, forceField.v().data());
    hermes::ThreadArrayDistributionInfo td(velocity.v().resolution());
    __applyForceField<<<td.gridSize, td.blockSize>>>(
        velocity.v().accessor(), forceField.v().accessor(), dt);
  }
  {
    memcpy(forceArray, forceField.w().data());
    hermes::ThreadArrayDistributionInfo td(velocity.w().resolution());
    __applyForceField<<<td.gridSize, td.blockSize>>>(
        velocity.w().accessor(), forceField.w().accessor(), dt);
  }
  hipUnbindTexture(forceTex3);
}

__global__ void __applyBuoyancyForceField(
    RegularGrid3Accessor<float> fu, RegularGrid3Accessor<float> fv,
    RegularGrid3Accessor<float> fw, RegularGrid3Accessor<float> density,
    RegularGrid3Accessor<float> temperature, float tamb, float alpha,
    float beta) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (fu.isIndexStored(x, y, z)) {
    auto wp = fu.worldPosition(x, y, z);
    auto p = temperature.gridPosition(wp) + vec3f(0.5f);
    // printf("%f %f %f (%f %f %f)= %f %f\n", wp.x, wp.y, wp.z, p.x, p.y, p.z,
    //        density(x, y, z), temperature(x, y, z));
    fu(x, y, z) = fw(x, y, z) = 0.f;
    fv(x, y, z) += -alpha * tex3D(densityTex3, p.x, p.y, p.z) +
                   beta * (tex3D(temperatureTex3, p.x, p.y, p.z) - tamb);
  }
}

template <>
void computeBuoyancyForceField(VectorGrid3D &forceField,
                               RegularGrid3Df &density,
                               RegularGrid3Df &temperature,
                               float ambientTemperature, float alpha,
                               float beta) {
  temperatureTex3.filterMode = hipFilterModeLinear;
  temperatureTex3.normalized = 0;
  densityTex3.filterMode = hipFilterModeLinear;
  densityTex3.normalized = 0;
  Array3<float> tArray(temperature.resolution());
  Array3<float> dArray(density.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(
      hipBindTextureToArray(temperatureTex3, tArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(densityTex3, dArray.data(), channelDesc));
  memcpy(tArray, temperature.data());
  memcpy(dArray, density.data());
  hermes::ThreadArrayDistributionInfo td(forceField.resolution());
  __applyBuoyancyForceField<<<td.gridSize, td.blockSize>>>(
      forceField.u().accessor(), forceField.v().accessor(),
      forceField.w().accessor(), density.accessor(), temperature.accessor(),
      ambientTemperature, alpha, beta);
  hipUnbindTexture(temperatureTex3);
  hipUnbindTexture(densityTex3);
}

__device__ float u_ijk(float i, float j, float k) {
  return (tex3D(uTex3, i + 1, j, k) - tex3D(uTex3, i, j, k)) / 2.f;
}
__device__ float v_ijk(float i, float j, float k) {
  return (tex3D(vTex3, i, j + 1, k) - tex3D(vTex3, i, j, k)) / 2.f;
}
__device__ float w_ijk(float i, float j, float k) {
  return (tex3D(wTex3, i, j, k + 1) - tex3D(wTex3, i, j, k)) / 2.f;
}

__global__ void __computeVorticity(RegularGrid3Accessor<float> wu,
                                   RegularGrid3Accessor<float> wv,
                                   RegularGrid3Accessor<float> ww) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (wu.isIndexStored(x, y, z)) {
    float i = x + 0.5;
    float j = y + 0.5;
    float k = z + 0.5;
    float inv = 1.f / (2.f * wu.spacing().x);
    wu(x, y, z) = (w_ijk(i, j + 1, k) - w_ijk(i, j - 1, k) -
                   v_ijk(i, j, k + 1) + v_ijk(i, j, k - 1)) *
                  inv;
    inv = 1.f / (2.f * wu.spacing().y);
    wv(x, y, z) = (u_ijk(i, j, k + 1) - u_ijk(i, j, k - 1) -
                   w_ijk(i + 1, j, k) + w_ijk(i - 1, j, k)) *
                  inv;
    inv = 1.f / (2.f * wu.spacing().z);
    ww(x, y, z) = (v_ijk(i + 1, j, k) - v_ijk(i - 1, j, k) -
                   u_ijk(i, j + 1, k) + u_ijk(i, j - 1, k)) *
                  inv;
  }
}

template <>
void computeVorticity(StaggeredGrid3D &velocity, RegularGrid3Duc &solid,
                      VectorGrid3D &vorticityField) {
  uTex3.filterMode = hipFilterModePoint;
  uTex3.normalized = 0;
  vTex3.filterMode = hipFilterModePoint;
  vTex3.normalized = 0;
  wTex3.filterMode = hipFilterModePoint;
  wTex3.normalized = 0;
  solidTex3.filterMode = hipFilterModePoint;
  solidTex3.normalized = 0;
  Array3<float> uArray(velocity.u().resolution());
  Array3<float> vArray(velocity.v().resolution());
  Array3<float> wArray(velocity.w().resolution());
  Array3<unsigned char> sArray(solid.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(uTex3, uArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(vTex3, vArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wTex3, wArray.data(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex3, sArray.data(), channelDesc));
  memcpy(uArray, velocity.u().data());
  memcpy(vArray, velocity.v().data());
  memcpy(wArray, velocity.w().data());
  memcpy(sArray, solid.data());
  hermes::ThreadArrayDistributionInfo td(vorticityField.resolution());
  __computeVorticity<<<td.gridSize, td.blockSize>>>(
      vorticityField.u().accessor(), vorticityField.v().accessor(),
      vorticityField.w().accessor());
  hipUnbindTexture(uTex3);
  hipUnbindTexture(vTex3);
  hipUnbindTexture(wTex3);
  hipUnbindTexture(solidTex3);
}

__global__ void __computeVorticityNorm(RegularGrid3Accessor<float> n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (n.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    if (tex3D(solidTex3, xc, yc, zc)) {
      n(x, y, z) = 0.f;
      return;
    }
    vec3f w(tex3D(wuTex3, xc, yc, zc), tex3D(wvTex3, xc, yc, zc),
            tex3D(wwTex3, xc, yc, zc));
    // n(x, y, z) = w.length();
  }
}

void computeVorticityNorm(RegularGrid3Duc &solid, VectorGrid3D &vorticityField,
                          RegularGrid3Df &wNorm) {
  wuTex3.filterMode = hipFilterModePoint;
  wuTex3.normalized = 0;
  wvTex3.filterMode = hipFilterModePoint;
  wvTex3.normalized = 0;
  wwTex3.filterMode = hipFilterModePoint;
  wwTex3.normalized = 0;
  solidTex3.filterMode = hipFilterModePoint;
  solidTex3.normalized = 0;
  Array3<float> wuArray(vorticityField.u().resolution());
  Array3<float> wvArray(vorticityField.v().resolution());
  Array3<float> wwArray(vorticityField.w().resolution());
  Array3<unsigned char> sArray(solid.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(wuTex3, wuArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wvTex3, wvArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wwTex3, wwArray.data(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex3, sArray.data(), channelDesc));
  memcpy(wuArray, vorticityField.u().data());
  memcpy(wvArray, vorticityField.v().data());
  memcpy(wwArray, vorticityField.w().data());
  memcpy(sArray, solid.data());
  hermes::ThreadArrayDistributionInfo td(vorticityField.resolution());
  __computeVorticityNorm<<<td.gridSize, td.blockSize>>>(wNorm.accessor());
  hipUnbindTexture(wuTex3);
  hipUnbindTexture(wvTex3);
  hipUnbindTexture(wwTex3);
  hipUnbindTexture(solidTex3);
}

__global__ void __computeVorticityComfinementForce(
    RegularGrid3Accessor<float> fu, RegularGrid3Accessor<float> fv,
    RegularGrid3Accessor<float> fw, RegularGrid3Accessor<float> wNorm,
    float eta) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  if (wNorm.isIndexStored(i, j, k)) {
    float xc = i + 0.5;
    float yc = j + 0.5;
    float zc = k + 0.5;
    if (tex3D(solidTex3, xc, yc, zc)) {
      fu(i, j, k) = 0.f;
      fv(i, j, k) = 0.f;
      fw(i, j, k) = 0.f;
      return;
    }
    // wNorm gradient
    vec3f inv(1.f / (2 * wNorm.spacing().x), 1.f / (2 * wNorm.spacing().y),
              1.f / (2 * wNorm.spacing().z));
    vec3f g((wNorm(i + 1, j, k) - wNorm(i - 1, j, k)) * inv.x,
            (wNorm(i, j + 1, k) - wNorm(i, j - 1, k)) * inv.y,
            (wNorm(i, j, k + 1) - wNorm(i, j, k - 1)) * inv.z);
    // normalized
    vec3f N = g / (float)(g.length() + 1e-20 * wNorm.spacing().x);
    vec3f w(tex3D(wuTex3, xc, yc, zc), tex3D(wvTex3, xc, yc, zc),
            tex3D(wwTex3, xc, yc, zc));
    vec3f force = cross(N, w);
    fu(i, j, k) = eta * wNorm.spacing().x * force.x;
    fu(i, j, k) = eta * wNorm.spacing().y * force.y;
    fu(i, j, k) = eta * wNorm.spacing().z * force.z;
  }
}

template <>
void computeVorticityConfinementForceField(StaggeredGrid3D &velocity,
                                           RegularGrid3Duc &solid,
                                           VectorGrid3D &vorticityField,
                                           VectorGrid3D &forceField, float eta,
                                           float dt) {
  computeVorticity(velocity, solid, vorticityField);
  // std::cerr << "VORTICITY U\n";
  // std::cerr << vorticityField.u().data() << std::endl;
  // std::cerr << "VORTICITY V\n";
  // std::cerr << vorticityField.v().data() << std::endl;
  // std::cerr << "VORTICITY W\n";
  // std::cerr << vorticityField.w().data() << std::endl;
  RegularGrid3Df wNorm(solid.resolution());
  computeVorticityNorm(solid, vorticityField, wNorm);
  // compute force
  RegularGrid3Df confForce(solid.resolution());
  wuTex3.filterMode = hipFilterModePoint;
  wuTex3.normalized = 0;
  wvTex3.filterMode = hipFilterModePoint;
  wvTex3.normalized = 0;
  wwTex3.filterMode = hipFilterModePoint;
  wwTex3.normalized = 0;
  solidTex3.filterMode = hipFilterModePoint;
  solidTex3.normalized = 0;
  Array3<float> wuArray(vorticityField.u().resolution());
  Array3<float> wvArray(vorticityField.v().resolution());
  Array3<float> wwArray(vorticityField.w().resolution());
  Array3<unsigned char> sArray(solid.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(wuTex3, wuArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wvTex3, wvArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wwTex3, wwArray.data(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex3, sArray.data(), channelDesc));
  memcpy(wuArray, vorticityField.u().data());
  memcpy(wvArray, vorticityField.v().data());
  memcpy(wwArray, vorticityField.w().data());
  memcpy(sArray, solid.data());
  hermes::ThreadArrayDistributionInfo td(vorticityField.resolution());
  __computeVorticityComfinementForce<<<td.gridSize, td.blockSize>>>(
      forceField.u().accessor(), forceField.v().accessor(),
      forceField.w().accessor(), wNorm.accessor(), eta);
  hipUnbindTexture(wuTex3);
  hipUnbindTexture(wvTex3);
  hipUnbindTexture(wwTex3);
  hipUnbindTexture(solidTex3);
}

__global__ void __computeDivergence(RegularGrid3Accessor<float> divergence,
                                    vec3f invdx) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (divergence.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    float left = tex3D(uTex3, xc, yc, zc);
    float right = tex3D(uTex3, xc + 1, yc, zc);
    float bottom = tex3D(vTex3, xc, yc, zc);
    float top = tex3D(vTex3, xc, yc + 1, zc);
    float back = tex3D(wTex3, xc, yc, zc);
    float front = tex3D(wTex3, xc, yc, zc + 1);
    unsigned char sleft = tex3D(solidTex3, xc - 1, yc, zc);
    unsigned char sright = tex3D(solidTex3, xc + 1, yc, zc);
    unsigned char sbottom = tex3D(solidTex3, xc, yc - 1, zc);
    unsigned char stop = tex3D(solidTex3, xc, yc + 1, zc);
    unsigned char sback = tex3D(solidTex3, xc, yc, zc - 1);
    unsigned char sfront = tex3D(solidTex3, xc, yc, zc + 1);
    if (sleft)
      left = 0; // tex3D(uSolidTex3, xc, yc, zc);
    if (sright)
      right = 0; // tex3D(uSolidTex3, xc + 1, yc, zc);
    if (sbottom)
      bottom = 0; // tex3D(vSolidTex3, xc, yc, zc);
    if (stop)
      top = 0; // tex3D(vSolidTex3, xc, yc + 1, zc);
    if (sback)
      back = 0; // tex3D(wSolidTex3, xc, yc, zc);
    if (sfront)
      front = 0; // tex3D(wSolidTex3, xc, yc, zc + 1);
    divergence(x, y, z) =
        dot(invdx, vec3f(right - left, top - bottom, front - back));
  }
}

template <>
void computeDivergence(
    StaggeredGrid3D &velocity,
    RegularGrid3<MemoryLocation::DEVICE, unsigned char> &solid,
    RegularGrid3Df &divergence) {
  uTex3.filterMode = hipFilterModePoint;
  uTex3.normalized = 0;
  vTex3.filterMode = hipFilterModePoint;
  vTex3.normalized = 0;
  wTex3.filterMode = hipFilterModePoint;
  wTex3.normalized = 0;
  solidTex3.filterMode = hipFilterModePoint;
  solidTex3.normalized = 0;
  Array3<float> uArray(velocity.u().resolution());
  Array3<float> vArray(velocity.v().resolution());
  Array3<float> wArray(velocity.w().resolution());
  Array3<unsigned char> sArray(solid.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(uTex3, uArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(vTex3, vArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wTex3, wArray.data(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex3, sArray.data(), channelDesc));
  memcpy(uArray, velocity.u().data());
  memcpy(vArray, velocity.v().data());
  memcpy(wArray, velocity.w().data());
  memcpy(sArray, solid.data());
  auto info = divergence.info();
  vec3f inv(-1.f / divergence.spacing().x);
  hermes::ThreadArrayDistributionInfo td(divergence.resolution());
  __computeDivergence<<<td.gridSize, td.blockSize>>>(divergence.accessor(),
                                                     inv);
  hipUnbindTexture(uTex3);
  hipUnbindTexture(vTex3);
  hipUnbindTexture(wTex3);
  hipUnbindTexture(solidTex3);
}

__global__ void __fillPressureMatrix(MemoryBlock3Accessor<FDMatrix3Entry> A,
                                     RegularGrid3Accessor<unsigned char> solid,
                                     float scale) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  if (A.isIndexValid(i, j, k)) {
    if (solid(i, j, k))
      return;
    A(i, j, k).diag = 0;
    A(i, j, k).x = 0;
    A(i, j, k).y = 0;
    A(i, j, k).z = 0;
    // left - right
    if (solid.isIndexStored(i - 1, j, k) && !solid(i - 1, j, k))
      A(i, j, k).diag += scale;
    if (solid.isIndexStored(i + 1, j, k)) {
      if (!solid(i + 1, j, k)) {
        A(i, j, k).diag += scale;
        A(i, j, k).x = -scale;
      } // else // EMPTY
      //   A(i, j, k).diag += scale;
    }
    // bottom - top
    if (solid.isIndexStored(i, j - 1, k) && !solid(i, j - 1, k))
      A(i, j, k).diag += scale;
    if (solid.isIndexStored(i, j + 1, k)) {
      if (!solid(i, j + 1, k)) {
        A(i, j, k).diag += scale;
        A(i, j, k).y = -scale;
      } // else // EMPTY
      //   A(i, j, k).diag += scale;
    }
    // back - front
    if (solid.isIndexStored(i, j, k - 1) && !solid(i, j, k - 1))
      A(i, j, k).diag += scale;
    if (solid.isIndexStored(i, j, k + 1)) {
      if (!solid(i, j, k + 1)) {
        A(i, j, k).diag += scale;
        A(i, j, k).z = -scale;
      } //  else // EMPTY
        //   A(i, j, k).diag += scale;
    }
  }
}

__global__ void __buildRHS(MemoryBlock3Accessor<int> indices,
                           RegularGrid3Accessor<float> divergence,
                           MemoryBlock1Accessor<double> rhs) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  if (indices.isIndexValid(i, j, k)) {
    if (indices(i, j, k) >= 0)
      rhs[indices(i, j, k)] = divergence(i, j, k);
  }
}

__global__ void __1To3(MemoryBlock3Accessor<int> indices,
                       MemoryBlock1Accessor<double> v,
                       MemoryBlock3Accessor<float> m) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  if (indices.isIndexValid(i, j, k)) {
    if (indices(i, j, k) >= 0)
      m(i, j, k) = v[indices(i, j, k)];
  }
}

template <>
size_t setupPressureSystem(RegularGrid3Df &divergence, RegularGrid3Duc &solid,
                           FDMatrix3D &pressureMatrix, float dt,
                           MemoryBlock1Dd &rhs) {
  // fill matrix
  float scale = dt / (divergence.spacing().x * divergence.spacing().x);
  hermes::ThreadArrayDistributionInfo td(divergence.resolution());
  __fillPressureMatrix<<<td.gridSize, td.blockSize>>>(
      pressureMatrix.dataAccessor(), solid.accessor(), scale);
  // compute indices
  auto res = divergence.resolution();
  MemoryBlock3<MemoryLocation::HOST, int> h_indices(res);
  h_indices.allocate();
  MemoryBlock3<MemoryLocation::HOST, unsigned char> h_solid(res);
  h_solid.allocate();
  memcpy(h_solid, solid.data());
  auto solidAcc = h_solid.accessor();
  auto indicesAcc = h_indices.accessor();
  int curIndex = 0;
  for (size_t k = 0; k < res.z; k++)
    for (size_t j = 0; j < res.y; j++)
      for (size_t i = 0; i < res.x; i++)
        if (!solidAcc(i, j, k)) {
          indicesAcc(i, j, k) = curIndex++;
        } else
          indicesAcc(i, j, k) = -1;
  memcpy(pressureMatrix.indexData(), h_indices);
  // rhs
  rhs.resize(curIndex);
  rhs.allocate();
  __buildRHS<<<td.gridSize, td.blockSize>>>(pressureMatrix.indexDataAccessor(),
                                            divergence.accessor(),
                                            rhs.accessor());
  return curIndex;
}

template <>
void solvePressureSystem(
    FDMatrix3D &A, RegularGrid3Df &divergence, RegularGrid3Df &pressure,
    RegularGrid3<MemoryLocation::DEVICE, unsigned char> &solid, float dt) {
  // setup system
  MemoryBlock1Dd rhs;
  setupPressureSystem(divergence, solid, A, dt, rhs);
  // apply incomplete Cholesky preconditioner
  // solve system
  MemoryBlock1Dd x(rhs.size(), 0.f);
  // FDMatrix3H H(A.gridSize());
  // H.copy(A);
  // auto acc = H.accessor();
  // std::cerr << acc << "rhs\n" << rhs << std::endl;
  std::cerr << "solve\n";
  pcg(x, A, rhs, rhs.size(), 1e-6);
  // std::cerr << residual << "\n" << x << std::endl;
  // store pressure values
  MemoryBlock1Dd sol(rhs.size(), 0);
  mul(A, x, sol);
  sub(sol, rhs, sol);
  std::cerr << "test solution " << std::endl;
  if (infnorm(sol, sol) > 1e-6)
    std::cerr << "WRONG PCG!\n";
  // std::cerr << sol << std::endl;
  hermes::ThreadArrayDistributionInfo td(pressure.resolution());
  __1To3<<<td.gridSize, td.blockSize>>>(A.indexDataAccessor(), x.accessor(),
                                        pressure.data().accessor());
  // std::cerr << pressure.data() << std::endl;
  // MemoryBlock1Hd h_sol(rhs.size(), 0);
  // memcpy(h_sol, sol);
  // auto hsol = h_sol.accessor();
  // MemoryBlock1Hd h_rhs(rhs.size(), 0);
  // memcpy(h_rhs, rhs);
  // auto hrhs = h_rhs.accessor();
  // for (int i = 0; i < rhs.size(); i++)
  //   if (fabs(hrhs[i] - hsol[i]) > 1e-6)
  //     std::cerr << "WRONG PCG!\n";
}

__global__ void __projectionStepU(RegularGrid3Accessor<float> u, float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (u.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    if (tex3D(solidTex3, xc - 1, yc, zc))
      u(x, y, z) = 0; // tex3D(uSolidTex3, xc - 1, yc, zc);
    else if (tex3D(solidTex3, xc, yc, zc))
      u(x, y, z) = 0; // tex3D(uSolidTex3, xc, yc, zc);
    else {
      float l = tex3D(pressureTex3, xc - 1, yc, zc);
      float r = tex3D(pressureTex3, xc, yc, zc);
      u(x, y, z) -= scale * (r - l);
    }
  }
}

__global__ void __projectionStepV(RegularGrid3Accessor<float> v, float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (v.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    if (tex3D(solidTex3, xc, yc - 1, zc))
      v(x, y, z) = 0; // tex3D(vSolidTex3, xc, yc - 1, zc);
    else if (tex3D(solidTex3, xc, yc, zc))
      v(x, y, z) = 0; // tex3D(vSolidTex3, xc, yc, zc);
    else {
      float b = tex3D(pressureTex3, xc, yc - 1, zc);
      float t = tex3D(pressureTex3, xc, yc, zc);
      v(x, y, z) -= scale * (t - b);
    }
  }
}

__global__ void __projectionStepW(RegularGrid3Accessor<float> w, float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (w.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    if (tex3D(solidTex3, xc, yc, zc - 1))
      w(x, y, z) = 0; // tex3D(wSolidTex3, xc, yc, zc - 1);
    else if (tex3D(solidTex3, xc, yc, zc))
      w(x, y, z) = 0; // tex3D(wSolidTex3, xc, yc, zc);
    else {
      float b = tex3D(pressureTex3, xc, yc, zc - 1);
      float f = tex3D(pressureTex3, xc, yc, zc);
      w(x, y, z) -= scale * (f - b);
    }
  }
}

template <>
void projectionStep(RegularGrid3Df &pressure, RegularGrid3Duc &solid,
                    StaggeredGrid3D &velocity, float dt) {
  pressureTex3.filterMode = hipFilterModePoint;
  pressureTex3.normalized = 0;
  solidTex3.filterMode = hipFilterModePoint;
  solidTex3.normalized = 0;
  Array3<float> pArray(pressure.resolution());
  Array3<unsigned char> sArray(solid.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(pressureTex3, pArray.data(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex3, sArray.data(), channelDesc));
  memcpy(pArray, pressure.data());
  memcpy(sArray, solid.data());
  {
    auto info = velocity.u().info();
    float invdx = 1.0 / info.spacing.x;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepU<<<td.gridSize, td.blockSize>>>(velocity.u().accessor(),
                                                     scale);
  }
  {
    auto info = velocity.v().info();
    float invdx = 1.0 / info.spacing.y;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepV<<<td.gridSize, td.blockSize>>>(velocity.v().accessor(),
                                                     scale);
  }
  {
    auto info = velocity.w().info();
    float invdx = 1.0 / info.spacing.z;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepW<<<td.gridSize, td.blockSize>>>(velocity.w().accessor(),
                                                     scale);
  }
  hipUnbindTexture(pressureTex3);
  hipUnbindTexture(solidTex3);
}

} // namespace cuda

} // namespace poseidon
