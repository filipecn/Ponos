#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 FilipeCN
 *
 * The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * iM the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

#include <poseidon/math/cuda_pcg.h>
#include <poseidon/solvers/cuda_smoke_solver3_kernels.h>

namespace poseidon {

namespace cuda {

texture<float, hipTextureType3D> uTex3, uCopyTex3;
texture<float, hipTextureType3D> vTex3, vCopyTex3;
texture<float, hipTextureType3D> wTex3, wCopyTex3;
texture<float, hipTextureType3D> densityTex3;
texture<float, hipTextureType3D> pressureTex3;
texture<float, hipTextureType3D> divergenceTex3;
texture<unsigned char, hipTextureType3D> solidTex3;
texture<float, hipTextureType3D> uSolidTex3;
texture<float, hipTextureType3D> vSolidTex3;
texture<float, hipTextureType3D> wSolidTex3;
texture<float, hipTextureType3D> forceTex3;
texture<float, hipTextureType3D> temperatureTex3;

using namespace hermes::cuda;

__global__ void __injectTemperature(RegularGrid3Accessor<float> t,
                                    RegularGrid3Accessor<float> tTarget,
                                    float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (t.isIndexStored(x, y, z))
    t(x, y, z) += (1 - expf(-dt)) * (tTarget(x, y, z) - t(x, y, z));
}

template <>
void injectTemperature(
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, float> &temperature,
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, float>
        &targetTemperature,
    float dt) {
  hermes::ThreadArrayDistributionInfo td(temperature.resolution());
  __injectTemperature<<<td.gridSize, td.blockSize>>>(
      temperature.accessor(), targetTemperature.accessor(), dt);
}

__global__ void __injectSmoke(RegularGrid3Accessor<float> s,
                              RegularGrid3Accessor<unsigned char> source,
                              float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (s.isIndexStored(x, y, z)) {
    s(x, y, z) += dt * source(x, y, z);
    s(x, y, z) = fminf(1.f, s(x, y, z));
  }
}

template <>
void injectSmoke(
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, float> &smoke,
    hermes::cuda::RegularGrid3<MemoryLocation::DEVICE, unsigned char> &source,
    float dt) {
  hermes::ThreadArrayDistributionInfo td(smoke.resolution());
  __injectSmoke<<<td.gridSize, td.blockSize>>>(smoke.accessor(),
                                               source.accessor(), dt);
}

__global__ void __applyForceField(RegularGrid3Accessor<float> velocity,
                                  RegularGrid3Accessor<float> force, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (velocity.isIndexStored(x, y, z)) {
    auto wp = velocity.worldPosition(x, y, z);
    auto p = force.gridPosition(wp) + vec3f(0.5f);
    velocity(x, y, z) += dt * tex3D(forceTex3, p.x, p.y, p.z);
  }
}

template <>
void applyForceField(StaggeredGrid3D &velocity, VectorGrid3D &forceField,
                     float dt) {
  Array3<float> forceArray(forceField.u().resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(forceTex3, forceArray.data(), channelDesc));
  {
    memcpy(forceArray, forceField.u().data());
    hermes::ThreadArrayDistributionInfo td(velocity.u().resolution());
    __applyForceField<<<td.gridSize, td.blockSize>>>(
        velocity.u().accessor(), forceField.u().accessor(), dt);
  }
  {
    memcpy(forceArray, forceField.v().data());
    hermes::ThreadArrayDistributionInfo td(velocity.v().resolution());
    __applyForceField<<<td.gridSize, td.blockSize>>>(
        velocity.v().accessor(), forceField.v().accessor(), dt);
  }
  {
    memcpy(forceArray, forceField.w().data());
    hermes::ThreadArrayDistributionInfo td(velocity.w().resolution());
    __applyForceField<<<td.gridSize, td.blockSize>>>(
        velocity.w().accessor(), forceField.w().accessor(), dt);
  }
  hipUnbindTexture(forceTex3);
}

__global__ void
__applyBuoyancyForceField(RegularGrid3Accessor<float> velocity,
                          RegularGrid3Accessor<float> temperature, float tamb,
                          float alpha, float beta, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (velocity.isIndexStored(x, y, z)) {
    auto wp = velocity.worldPosition(x, y, z);
    auto p = temperature.gridPosition(wp) + vec3f(0.5f);
    velocity(x, y, z) +=
        dt * (-alpha * tex3D(densityTex3, p.x, p.y, p.z) +
              beta * (tex3D(temperatureTex3, p.x, p.y, p.z) - tamb));
  }
}

template <>
void applyBuoyancyForceField(StaggeredGrid3D &velocity, RegularGrid3Df &density,
                             RegularGrid3Df &temperature,
                             float ambientTemperature, float alpha, float beta,
                             float dt) {
  Array3<float> tArray(temperature.resolution());
  Array3<float> dArray(density.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(
      hipBindTextureToArray(temperatureTex3, tArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(densityTex3, dArray.data(), channelDesc));
  memcpy(tArray, temperature.data());
  memcpy(dArray, density.data());
  hermes::ThreadArrayDistributionInfo td(velocity.v().resolution());
  __applyBuoyancyForceField<<<td.gridSize, td.blockSize>>>(
      velocity.v().accessor(), temperature.accessor(), ambientTemperature,
      alpha, beta, dt);
  hipUnbindTexture(temperatureTex3);
  hipUnbindTexture(densityTex3);
}

__global__ void __computeDivergence(RegularGrid3Accessor<float> divergence,
                                    vec3f invdx) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (divergence.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    float left = tex3D(uTex3, xc, yc, zc);
    float right = tex3D(uTex3, xc + 1, yc, zc);
    float bottom = tex3D(vTex3, xc, yc, zc);
    float top = tex3D(vTex3, xc, yc + 1, zc);
    float back = tex3D(wTex3, xc, yc, zc);
    float front = tex3D(wTex3, xc, yc, zc + 1);
    unsigned char sleft = tex3D(solidTex3, xc - 1, yc, zc);
    unsigned char sright = tex3D(solidTex3, xc + 1, yc, zc);
    unsigned char sbottom = tex3D(solidTex3, xc, yc - 1, zc);
    unsigned char stop = tex3D(solidTex3, xc, yc + 1, zc);
    unsigned char sback = tex3D(solidTex3, xc, yc, zc - 1);
    unsigned char sfront = tex3D(solidTex3, xc, yc, zc + 1);
    if (sleft)
      left = 0; // tex3D(uSolidTex3, xc, yc, zc);
    if (sright)
      right = 0; // tex3D(uSolidTex3, xc + 1, yc, zc);
    if (sbottom)
      bottom = 0; // tex3D(vSolidTex3, xc, yc, zc);
    if (stop)
      top = 0; // tex3D(vSolidTex3, xc, yc + 1, zc);
    if (sback)
      back = 0; // tex3D(wSolidTex3, xc, yc, zc);
    if (sfront)
      front = 0; // tex3D(wSolidTex3, xc, yc, zc + 1);
    divergence(x, y, z) =
        dot(invdx, vec3f(right - left, top - bottom, front - back));
  }
}

template <>
void computeDivergence(
    StaggeredGrid3D &velocity,
    RegularGrid3<MemoryLocation::DEVICE, unsigned char> &solid,
    RegularGrid3Df &divergence) {
  Array3<float> uArray(velocity.u().resolution());
  Array3<float> vArray(velocity.v().resolution());
  Array3<float> wArray(velocity.w().resolution());
  Array3<unsigned char> sArray(solid.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(uTex3, uArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(vTex3, vArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wTex3, wArray.data(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex3, sArray.data(), channelDesc));
  memcpy(uArray, velocity.u().data());
  memcpy(vArray, velocity.v().data());
  memcpy(wArray, velocity.w().data());
  memcpy(sArray, solid.data());
  auto info = divergence.info();
  vec3f inv(1.f / divergence.spacing().x);
  hermes::ThreadArrayDistributionInfo td(divergence.resolution());
  __computeDivergence<<<td.gridSize, td.blockSize>>>(divergence.accessor(),
                                                     inv);
  hipUnbindTexture(uTex3);
  hipUnbindTexture(vTex3);
  hipUnbindTexture(wTex3);
  hipUnbindTexture(solidTex3);
}

__global__ void __fillPressureMatrix(MemoryBlock3Accessor<FDMatrix3Entry> A,
                                     RegularGrid3Accessor<unsigned char> solid,
                                     float scale) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  if (A.isIndexValid(i, j, k)) {
    if (solid(i, j, k))
      return;
    A(i, j, k).diag = 0;
    A(i, j, k).x = 0;
    A(i, j, k).y = 0;
    A(i, j, k).z = 0;
    // left - right
    if (solid.isIndexStored(i - 1, j, k) && !solid(i - 1, j, k))
      A(i, j, k).diag += scale;
    if (solid.isIndexStored(i + 1, j, k)) {
      if (!solid(i + 1, j, k)) {
        A(i, j, k).diag += scale;
        A(i, j, k).x = -scale;
      } // else // EMPTY
      //   A(i, j, k).diag += scale;
    }
    // bottom - top
    if (solid.isIndexStored(i, j - 1, k) && !solid(i, j - 1, k))
      A(i, j, k).diag += scale;
    if (solid.isIndexStored(i, j + 1, k)) {
      if (!solid(i, j + 1, k)) {
        A(i, j, k).diag += scale;
        A(i, j, k).y = -scale;
      } // else // EMPTY
      //   A(i, j, k).diag += scale;
    }
    // back - front
    if (solid.isIndexStored(i, j, k - 1) && !solid(i, j, k - 1))
      A(i, j, k).diag += scale;
    if (solid.isIndexStored(i, j, k + 1)) {
      if (!solid(i, j, k + 1)) {
        A(i, j, k).diag += scale;
        A(i, j, k).z = -scale;
      } //  else // EMPTY
        //   A(i, j, k).diag += scale;
    }
  }
}

__global__ void __buildRHS(MemoryBlock3Accessor<int> indices,
                           RegularGrid3Accessor<float> divergence,
                           MemoryBlock1Accessor<float> rhs) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  if (indices.isIndexValid(i, j, k)) {
    if (indices(i, j, k) >= 0)
      rhs[indices(i, j, k)] = divergence(i, j, k);
  }
}

__global__ void __1To3(MemoryBlock3Accessor<int> indices,
                       MemoryBlock1Accessor<float> v,
                       MemoryBlock3Accessor<float> m) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  if (indices.isIndexValid(i, j, k)) {
    if (indices(i, j, k) >= 0)
      m(i, j, k) = v[indices(i, j, k)];
  }
}

template <>
size_t setupPressureSystem(RegularGrid3Df &divergence, RegularGrid3Duc &solid,
                           FDMatrix3D &pressureMatrix, float dt,
                           MemoryBlock1Df &rhs) {
  // fill matrix
  float scale = dt / (divergence.spacing().x * divergence.spacing().x);
  hermes::ThreadArrayDistributionInfo td(divergence.resolution());
  __fillPressureMatrix<<<td.gridSize, td.blockSize>>>(
      pressureMatrix.dataAccessor(), solid.accessor(), scale);
  // compute indices
  auto res = divergence.resolution();
  MemoryBlock3<MemoryLocation::HOST, int> h_indices(res);
  h_indices.allocate();
  MemoryBlock3<MemoryLocation::HOST, unsigned char> h_solid(res);
  h_solid.allocate();
  memcpy(h_solid, solid.data());
  auto solidAcc = h_solid.accessor();
  auto indicesAcc = h_indices.accessor();
  int curIndex = 0;
  for (size_t k = 0; k < res.z; k++)
    for (size_t j = 0; j < res.y; j++)
      for (size_t i = 0; i < res.x; i++)
        if (!solidAcc(i, j, k)) {
          indicesAcc(i, j, k) = curIndex++;
        } else
          indicesAcc(i, j, k) = -1;
  memcpy(pressureMatrix.indexData(), h_indices);
  // rhs
  rhs.resize(curIndex);
  rhs.allocate();
  __buildRHS<<<td.gridSize, td.blockSize>>>(pressureMatrix.indexDataAccessor(),
                                            divergence.accessor(),
                                            rhs.accessor());
  return curIndex;
}

template <>
void solvePressureSystem(
    FDMatrix3D &A, RegularGrid3Df &divergence, RegularGrid3Df &pressure,
    RegularGrid3<MemoryLocation::DEVICE, unsigned char> &solid, float dt) {
  // setup system
  MemoryBlock1Df rhs;
  setupPressureSystem(divergence, solid, A, dt, rhs);
  // apply incomplete Cholesky preconditioner
  // solve system
  MemoryBlock1Df x(rhs.size(), 0.f);
  float residual;
  pcg(x, A, rhs, 100, &residual);
  // store pressure values
  hermes::ThreadArrayDistributionInfo td(pressure.resolution());
  __1To3<<<td.gridSize, td.blockSize>>>(A.indexDataAccessor(), x.accessor(),
                                        pressure.data().accessor());
}

__global__ void __projectionStepU(RegularGrid3Accessor<float> u, float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (u.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    if (tex3D(solidTex3, xc - 1, yc, zc))
      u(x, y, z) = 0; // tex3D(uSolidTex3, xc - 1, yc, zc);
    else if (tex3D(solidTex3, xc, yc, zc))
      u(x, y, z) = 0; // tex3D(uSolidTex3, xc, yc, zc);
    else {
      float l = tex3D(pressureTex3, xc - 1, yc, zc);
      float r = tex3D(pressureTex3, xc, yc, zc);
      u(x, y, z) -= scale * (r - l);
    }
  }
}

__global__ void __projectionStepV(RegularGrid3Accessor<float> v, float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (v.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    if (tex3D(solidTex3, xc, yc - 1, zc))
      v(x, y, z) = 0; // tex3D(vSolidTex3, xc, yc - 1, zc);
    else if (tex3D(solidTex3, xc, yc, zc))
      v(x, y, z) = 0; // tex3D(vSolidTex3, xc, yc, zc);
    else {
      float b = tex3D(pressureTex3, xc, yc - 1, zc);
      float t = tex3D(pressureTex3, xc, yc, zc);
      v(x, y, z) -= scale * (t - b);
    }
  }
}

__global__ void __projectionStepW(RegularGrid3Accessor<float> w, float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (w.isIndexStored(x, y, z)) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float zc = z + 0.5;
    if (tex3D(solidTex3, xc, yc, zc - 1))
      w(x, y, z) = 0; // tex3D(wSolidTex3, xc, yc, zc - 1);
    else if (tex3D(solidTex3, xc, yc, zc))
      w(x, y, z) = 0; // tex3D(wSolidTex3, xc, yc, zc);
    else {
      float b = tex3D(pressureTex3, xc, yc, zc - 1);
      float f = tex3D(pressureTex3, xc, yc, zc);
      w(x, y, z) -= scale * (f - b);
    }
  }
}

template <>
void projectionStep(RegularGrid3Df &pressure, RegularGrid3Duc &solid,
                    StaggeredGrid3D &velocity, float dt) {
  Array3<float> pArray(pressure.resolution());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(pressureTex3, pArray.data(), channelDesc));
  memcpy(pArray, pressure.data());
  {
    auto info = velocity.u().info();
    float invdx = 1.0 / info.spacing.x;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepU<<<td.gridSize, td.blockSize>>>(velocity.u().accessor(),
                                                     scale);
  }
  {
    auto info = velocity.v().info();
    float invdx = 1.0 / info.spacing.y;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepV<<<td.gridSize, td.blockSize>>>(velocity.v().accessor(),
                                                     scale);
  }
  {
    auto info = velocity.w().info();
    float invdx = 1.0 / info.spacing.z;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepU<<<td.gridSize, td.blockSize>>>(velocity.w().accessor(),
                                                     scale);
  }
  hipUnbindTexture(pressureTex3);
}

} // namespace cuda

} // namespace poseidon
