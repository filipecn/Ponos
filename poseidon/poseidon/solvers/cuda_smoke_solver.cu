#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 FilipeCN
 *
 * The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * iM the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

#ifndef POSEIDON_SOLVERS_CUDA_SMOKE_SOLVER_KERNELS_H
#define POSEIDON_SOLVERS_CUDA_SMOKE_SOLVER_KERNELS_H

#include <hermes/common/hip/hip_runtime.h>
#include <poseidon/solvers/cuda_smoke_solver.h>

namespace poseidon {

namespace cuda {

texture<float, hipTextureType2D> uTex2;
texture<float, hipTextureType2D> vTex2;
texture<float, hipTextureType2D> densityTex2;
texture<float, hipTextureType2D> pressureTex2;
texture<float, hipTextureType2D> divergenceTex2;
texture<unsigned char, hipTextureType2D> solidTex2;
texture<float, hipTextureType2D> uSolidTex2;
texture<float, hipTextureType2D> vSolidTex2;

__global__ void __setupScene(poseidon::cuda::Collider2<float> **solids,
                             poseidon::cuda::Collider2<float> **scene) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    solids[0] = new poseidon::cuda::SphereCollider2<float>(
        hermes::cuda::point2(0.f, 0.f), 0.1f);
    float d = 1.0 / 64;
    // floor
    solids[1] = new poseidon::cuda::BoxCollider2<float>(hermes::cuda::bbox2(
        hermes::cuda::point2(0.f, 0.f), hermes::cuda::point2(1.f, d)));
    // ceil
    solids[2] = new poseidon::cuda::BoxCollider2<float>(hermes::cuda::bbox2(
        hermes::cuda::point2(0.f, 1.f - d), hermes::cuda::point2(1.f, 1.f)));
    // left
    solids[3] = new poseidon::cuda::BoxCollider2<float>(hermes::cuda::bbox2(
        hermes::cuda::point2(0.f, 0.f), hermes::cuda::point2(d, 1.f)));
    // right
    solids[4] = new poseidon::cuda::BoxCollider2<float>(hermes::cuda::bbox2(
        hermes::cuda::point2(1.f - d, 0.f), hermes::cuda::point2(1.f, 1.f)));
    *scene = new poseidon::cuda::Collider2Set<float>(solids, 5);
  }
}

__global__ void __freeScene(poseidon::cuda::Collider2<float> **solids) {
  if (threadIdx.x == 0 && blockIdx.x == 0)
    for (int i = 0; i < 5; ++i)
      delete solids[i];
}

__global__ void __rasterColliders(Collider2<float> *const *colliders,
                                  unsigned char *solids, float *u, float *v,
                                  hermes::cuda::Grid2Info sInfo) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * sInfo.resolution.x + x;
  if (x < sInfo.resolution.x && y < sInfo.resolution.y) {
    if ((*colliders)->intersect(sInfo.toWorld(hermes::cuda::point2(x, y))))
      solids[index] = 1;
    else
      solids[index] = 0;
    u[y * sInfo.resolution.x + x] = u[y * sInfo.resolution.x + x + 1] = 0;
    v[y * sInfo.resolution.x + x] = v[(y + 1) * sInfo.resolution.x + x] = 0;
  }
}

__global__ void __addGravityV(float *v, hermes::cuda::Grid2Info vInfo, float g,
                              float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * vInfo.resolution.x + x;
  if (x < vInfo.resolution.x && y < vInfo.resolution.y) {
    v[index] += dt * g;
  }
}

__global__ void __advectUVelocities(float *u, hermes::cuda::Grid2Info uInfo,
                                    hermes::cuda::Grid2Info vInfo, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * uInfo.resolution.x + x;
  if (x < uInfo.resolution.x && y < uInfo.resolution.y) {
    unsigned char solid = tex2D(solidTex2, x + 0.5, y + 0.5);
    if (solid) {
      u[index] = 0;
      return;
    }
    hermes::cuda::point2f wp = uInfo.toWorld(hermes::cuda::point2f(x, y));
    hermes::cuda::point2f vp = vInfo.toField(wp) + hermes::cuda::vec2(0.5);
    hermes::cuda::vec2f vel(tex2D(uTex2, x + 0.5, y + 0.5),
                            tex2D(vTex2, vp.x, vp.y));
    hermes::cuda::point2f pos = wp - vel * dt + hermes::cuda::vec2(0.5);
    u[index] = tex2D(uTex2, pos.x, pos.y);
  }
}

__global__ void __advectVVelocities(float *v, hermes::cuda::Grid2Info uInfo,
                                    hermes::cuda::Grid2Info vInfo, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * vInfo.resolution.x + x;
  if (x < vInfo.resolution.x && y < vInfo.resolution.y) {
    unsigned char solid = tex2D(solidTex2, x + 0.5, y + 0.5);
    if (solid) {
      v[index] = 0;
      return;
    }
    hermes::cuda::point2f wp = vInfo.toWorld(hermes::cuda::point2f(x, y));
    hermes::cuda::point2f up = uInfo.toField(wp) + hermes::cuda::vec2(0.5);
    hermes::cuda::vec2f vel(tex2D(uTex2, up.x, up.y),
                            tex2D(vTex2, x + 0.5, y + 0.5));
    hermes::cuda::point2f pos = wp - vel * dt + hermes::cuda::vec2(0.5);
    v[index] = tex2D(vTex2, pos.x, pos.y);
  }
}

__global__ void __advectDensity(float *d, hermes::cuda::Grid2Info dInfo,
                                hermes::cuda::Grid2Info uInfo,
                                hermes::cuda::Grid2Info vInfo, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * dInfo.resolution.x + x;
  if (x < dInfo.resolution.x && y < dInfo.resolution.y) {
    unsigned char solid = tex2D(solidTex2, x + 0.5, y + 0.5);
    if (solid) {
      d[index] = 0;
      return;
    }
    hermes::cuda::point2f wp = dInfo.toWorld(hermes::cuda::point2f(x, y));
    hermes::cuda::point2f up = uInfo.toField(wp) + hermes::cuda::vec2(0.5);
    hermes::cuda::point2f vp = vInfo.toField(wp) + hermes::cuda::vec2(0.5);
    hermes::cuda::vec2f vel(tex2D(uTex2, up.x, up.y), tex2D(vTex2, vp.x, vp.y));
    hermes::cuda::point2f pos =
        dInfo.toField(wp - vel * dt) + hermes::cuda::vec2(0.5);
    d[index] = tex2D(densityTex2, pos.x, pos.y);
  }
}

__global__ void __computeDivergence(float *d, hermes::cuda::Grid2Info dInfo,
                                    float invdx) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * dInfo.resolution.x + x;
  if (x < dInfo.resolution.x && y < dInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    unsigned char sl = tex2D(solidTex2, xc - 1, yc);
    unsigned char sr = tex2D(solidTex2, xc + 1, yc);
    unsigned char sb = tex2D(solidTex2, xc, yc - 1);
    unsigned char st = tex2D(solidTex2, xc, yc + 1);
    float l = tex2D(uTex2, xc, yc);
    float r = tex2D(uTex2, xc + 1, yc);
    float b = tex2D(vTex2, xc, yc);
    float t = tex2D(vTex2, xc, yc + 1);
    if (sl)
      l = tex2D(uSolidTex2, xc, yc);
    if (sr)
      r = tex2D(uSolidTex2, xc + 1, yc);
    if (sb)
      b = tex2D(vSolidTex2, xc, yc);
    if (st)
      t = tex2D(vSolidTex2, xc, yc + 1);

    d[index] = 0.5 * invdx * (t - b + r - l);
  }
}

__global__ void __computePressure(float *p, hermes::cuda::Grid2Info pInfo,
                                  float alpha, float beta) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * pInfo.resolution.x + x;
  if (x < pInfo.resolution.x && y < pInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float c = tex2D(pressureTex2, xc, yc);
    float l = tex2D(pressureTex2, xc - 1, yc);
    float r = tex2D(pressureTex2, xc + 1, yc);
    float b = tex2D(pressureTex2, xc, yc - 1);
    float t = tex2D(pressureTex2, xc, yc + 1);
    float rhs = tex2D(divergenceTex2, xc, yc);
    unsigned char sl = tex2D(solidTex2, xc - 1, yc);
    unsigned char sr = tex2D(solidTex2, xc + 1, yc);
    unsigned char sb = tex2D(solidTex2, xc, yc - 1);
    unsigned char st = tex2D(solidTex2, xc, yc + 1);
    if (sl)
      l = c;
    if (sr)
      r = c;
    if (sb)
      b = c;
    if (st)
      t = c;
    p[index] = (l + r + t + b + alpha * rhs) * beta;
  }
}

__global__ void __projectionStepU(float *u, hermes::cuda::Grid2Info uInfo,
                                  float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * uInfo.resolution.x + x;
  if (x < uInfo.resolution.x && y < uInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float l = tex2D(pressureTex2, xc - 1, yc);
    float r = tex2D(pressureTex2, xc, yc);
    u[index] -= 0.5 * scale * (r - l);
  }
}

__global__ void __projectionStepV(float *v, hermes::cuda::Grid2Info vInfo,
                                  float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * vInfo.resolution.x + x;
  if (x < vInfo.resolution.x && y < vInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float b = tex2D(pressureTex2, xc, yc - 1);
    float t = tex2D(pressureTex2, xc, yc);
    v[index] -= 0.5 * scale * (t - b);
  }
}

GridSmokeSolver2::~GridSmokeSolver2() {
  hipUnbindTexture(vTex2);
  hipUnbindTexture(uTex2);
  hipUnbindTexture(densityTex2);
  hipUnbindTexture(divergenceTex2);
  hipUnbindTexture(pressureTex2);
  hipUnbindTexture(solidTex2);
  hipUnbindTexture(uSolidTex2);
  hipUnbindTexture(vSolidTex2);

  __freeScene<<<1, 1>>>(scene.list);
  using namespace hermes::cuda;
  CUDA_CHECK(hipFree(scene.list));
  CUDA_CHECK(hipFree(scene.colliders));
}

void GridSmokeSolver2::setResolution(const ponos::uivec2 &res) {
  resolution = hermes::cuda::vec2u(res.x, res.y);
  velocity.resize(resolution);
  density.resize(resolution);
  pressure.resize(resolution);
  divergence.resize(resolution);
  solid.resize(resolution);
  solidVelocity.resize(resolution);
}

void GridSmokeSolver2::setDx(float _dx) {
  dx = _dx;
  velocity.setDx(dx);
  density.setDx(dx);
  pressure.setDx(dx);
  divergence.setDx(dx);
  solid.setDx(dx);
  solidVelocity.setDx(dx);
}

void GridSmokeSolver2::setOrigin(const ponos::point2f &o) {
  hermes::cuda::point2f p(o.x, o.y);
  velocity.setOrigin(p);
  density.setOrigin(p);
  pressure.setOrigin(p);
  divergence.setOrigin(p);
  solid.setOrigin(p);
  solidVelocity.setOrigin(p);
}

void GridSmokeSolver2::init() {
  setupTextures();
  {
    using namespace hermes::cuda;
    CUDA_CHECK(hipMalloc(&scene.list, 5 * sizeof(Collider2<float> *)));
    CUDA_CHECK(hipMalloc(&scene.colliders, sizeof(Collider2<float> *)));
  }
}

void GridSmokeSolver2::step(float dt) {
  rasterColliders();
  advectVelocities(dt);
  advectDensity(dt);
  // inject
  addGravity(dt);
  computeDivergence();
  computePressure();
  projectionStep(dt);
}

void GridSmokeSolver2::rasterColliders() {
  __setupScene<<<1, 1>>>(scene.list, scene.colliders);
  hermes::ThreadArrayDistributionInfo td(resolution.x, resolution.y);
  __rasterColliders<<<td.gridSize, td.blockSize>>>(
      scene.colliders, solid.texture().deviceData(),
      solidVelocity.uDeviceData(), solidVelocity.vDeviceData(), solid.info());
  solid.texture().updateTextureMemory();
}

void GridSmokeSolver2::setupTextures() {
  // uTex2.addressMode[0] = hipAddressModeBorder;
  // uTex2.addressMode[1] = hipAddressModeBorder;
  uSolidTex2.filterMode = hipFilterModeLinear;
  uSolidTex2.normalized = 0;
  vSolidTex2.filterMode = hipFilterModeLinear;
  vSolidTex2.normalized = 0;
  uTex2.filterMode = hipFilterModeLinear;
  uTex2.normalized = 0;
  vTex2.filterMode = hipFilterModeLinear;
  vTex2.normalized = 0;
  densityTex2.filterMode = hipFilterModeLinear;
  densityTex2.normalized = 0;
  divergenceTex2.filterMode = hipFilterModeLinear;
  divergenceTex2.normalized = 0;
  pressureTex2.filterMode = hipFilterModeLinear;
  pressureTex2.normalized = 0;
  solidTex2.filterMode = hipFilterModePoint;
  solidTex2.normalized = 0;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  using namespace hermes::cuda;
  CUDA_CHECK(hipBindTextureToArray(
      uTex2, velocity.u().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      vTex2, velocity.v().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      densityTex2, density.texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      divergenceTex2, divergence.texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      pressureTex2, pressure.texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      uSolidTex2, solidVelocity.u().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      vSolidTex2, solidVelocity.v().texture().textureArray(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex2, solid.texture().textureArray(),
                                    channelDesc));
}

void GridSmokeSolver2::addGravity(float dt) {
  hermes::ThreadArrayDistributionInfo td(resolution.x, resolution.y + 1);
  __addGravityV<<<td.gridSize, td.blockSize>>>(velocity.vDeviceData(),
                                               velocity.v().info(), -9.81f, dt);
  velocity.v().texture().updateTextureMemory();
}

void GridSmokeSolver2::advectVelocities(float dt) {
  {
    hermes::ThreadArrayDistributionInfo td(resolution.x + 1, resolution.y);
    __advectUVelocities<<<td.gridSize, td.blockSize>>>(
        velocity.uDeviceData(), velocity.u().info(), velocity.v().info(), dt);
  }
  {
    hermes::ThreadArrayDistributionInfo td(resolution.x, resolution.y + 1);
    __advectVVelocities<<<td.gridSize, td.blockSize>>>(
        velocity.vDeviceData(), velocity.u().info(), velocity.v().info(), dt);
  }
}

void GridSmokeSolver2::advectDensity(float dt) {
  hermes::ThreadArrayDistributionInfo td(resolution.x, resolution.y);
  __advectDensity<<<td.gridSize, td.blockSize>>>(
      density.texture().deviceData(), density.info(), velocity.u().info(),
      velocity.v().info(), dt);
  density.texture().updateTextureMemory();
}

void GridSmokeSolver2::computeDivergence() {
  float invdx = 1.0 / dx;
  hermes::ThreadArrayDistributionInfo td(resolution.x, resolution.y);
  __computeDivergence<<<td.gridSize, td.blockSize>>>(
      divergence.texture().deviceData(), divergence.info(), invdx);
  divergence.texture().updateTextureMemory();
}

void GridSmokeSolver2::computePressure() {
  hermes::ThreadArrayDistributionInfo td(resolution.x, resolution.y);
  float alpha = -1.0 / (dx * dx);
  float beta = 0.25;
  int iterations = 100;
  for (int i = 0; i < iterations; i++) {
    pressure.texture().updateTextureMemory();
    __computePressure<<<td.gridSize, td.blockSize>>>(
        pressure.texture().deviceData(), pressure.info(), alpha, beta);
  }
  pressure.texture().updateTextureMemory();
}

void GridSmokeSolver2::projectionStep(float dt) {
  float invdx = 1.0 / dx;
  float scale = dt * invdx;
  hermes::ThreadArrayDistributionInfo td(resolution.x, resolution.y);
  __projectionStepU<<<td.gridSize, td.blockSize>>>(velocity.uDeviceData(),
                                                   velocity.u().info(), scale);
  velocity.u().texture().updateTextureMemory();
  __projectionStepV<<<td.gridSize, td.blockSize>>>(velocity.vDeviceData(),
                                                   velocity.v().info(), scale);
  velocity.v().texture().updateTextureMemory();
}

hermes::cuda::StaggeredGridTexture2 &GridSmokeSolver2::velocityData() {
  return velocity;
}

const hermes::cuda::GridTexture2<float> &GridSmokeSolver2::densityData() const {
  return density;
}

const hermes::cuda::GridTexture2<unsigned char> &
GridSmokeSolver2::solidData() const {
  return solid;
}

hermes::cuda::GridTexture2<unsigned char> &GridSmokeSolver2::solidData() {
  return solid;
}

hermes::cuda::GridTexture2<float> &GridSmokeSolver2::densityData() {
  return density;
}

const hermes::cuda::StaggeredGridTexture2 &
GridSmokeSolver2::solidVelocityData() const {
  return solidVelocity;
}

hermes::cuda::StaggeredGridTexture2 &GridSmokeSolver2::solidVelocityData() {
  return solidVelocity;
}

} // namespace cuda

} // namespace poseidon

#endif // POSEIDON_SOLVERS_CUDA_SMOKE_SOLVER_KERNELS_H