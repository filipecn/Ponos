#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 FilipeCN
 *
 * The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * iM the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

#include <poseidon/solvers/cuda_smoke_solver_kernels.h>

namespace poseidon {

namespace cuda {

texture<float, hipTextureType2D> uTex2, uCopyTex2;
texture<float, hipTextureType2D> vTex2, vCopyTex2;
texture<float, hipTextureType2D> densityTex2;
texture<float, hipTextureType2D> pressureTex2;
texture<float, hipTextureType2D> divergenceTex2;
texture<unsigned char, hipTextureType2D> solidTex2;
texture<float, hipTextureType2D> uSolidTex2;
texture<float, hipTextureType2D> vSolidTex2;
texture<float, hipTextureType2D> uForceTex2;
texture<float, hipTextureType2D> vForceTex2;

__global__ void __applyForceFieldU(float *u, hermes::cuda::Grid2Info uInfo,
                                   float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * uInfo.resolution.x + x;
  if (x < uInfo.resolution.x && y < uInfo.resolution.y) {
    u[index] += dt * tex2D(uForceTex2, x + 0.5, y + 0.5);
  }
}

__global__ void __applyForceFieldV(float *v, hermes::cuda::Grid2Info vInfo,
                                   float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * vInfo.resolution.x + x;
  if (x < vInfo.resolution.x && y < vInfo.resolution.y) {
    v[index] += dt * tex2D(vForceTex2, x + 0.5, y + 0.5);
  }
}

__global__ void __advectDensity(float *d, hermes::cuda::Grid2Info dInfo,
                                hermes::cuda::Grid2Info uInfo,
                                hermes::cuda::Grid2Info vInfo, float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * dInfo.resolution.x + x;
  if (x < dInfo.resolution.x && y < dInfo.resolution.y) {
    unsigned char solid = tex2D(solidTex2, x + 0.5, y + 0.5);
    if (solid) {
      d[index] = 0;
      return;
    }
    hermes::cuda::point2f wp = dInfo.toWorld(hermes::cuda::point2f(x, y));
    hermes::cuda::point2f up = uInfo.toField(wp) + hermes::cuda::vec2(0.5);
    hermes::cuda::point2f vp = vInfo.toField(wp) + hermes::cuda::vec2(0.5);
    hermes::cuda::vec2f vel(tex2D(uTex2, up.x, up.y), tex2D(vTex2, vp.x, vp.y));
    hermes::cuda::point2f pos =
        dInfo.toField(wp - vel * dt) + hermes::cuda::vec2(0.5);
    d[index] = tex2D(densityTex2, pos.x, pos.y);
  }
}

__global__ void __computeDivergence(float *d, hermes::cuda::Grid2Info dInfo,
                                    float invdx) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * dInfo.resolution.x + x;
  if (x < dInfo.resolution.x && y < dInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float l = tex2D(uTex2, xc, yc);
    float r = tex2D(uTex2, xc + 1, yc);
    float b = tex2D(vTex2, xc, yc);
    float t = tex2D(vTex2, xc, yc + 1);
    unsigned char sl = tex2D(solidTex2, xc - 1, yc);
    unsigned char sr = tex2D(solidTex2, xc + 1, yc);
    unsigned char sb = tex2D(solidTex2, xc, yc - 1);
    unsigned char st = tex2D(solidTex2, xc, yc + 1);
    if (sl)
      l = tex2D(uSolidTex2, xc, yc);
    if (sr)
      r = tex2D(uSolidTex2, xc + 1, yc);
    if (sb)
      b = tex2D(vSolidTex2, xc, yc);
    if (st)
      t = tex2D(vSolidTex2, xc, yc + 1);
    d[index] = invdx * (t - b + r - l);
  }
}

__global__ void __computePressure(float *p, hermes::cuda::Grid2Info pInfo,
                                  float alpha) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * pInfo.resolution.x + x;
  if (x < pInfo.resolution.x && y < pInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float c = tex2D(pressureTex2, xc, yc);
    float l = tex2D(pressureTex2, xc - 1, yc);
    float r = tex2D(pressureTex2, xc + 1, yc);
    float b = tex2D(pressureTex2, xc, yc - 1);
    float t = tex2D(pressureTex2, xc, yc + 1);
    float rhs = tex2D(divergenceTex2, xc, yc);
    unsigned char sl = tex2D(solidTex2, xc - 1, yc);
    unsigned char sr = tex2D(solidTex2, xc + 1, yc);
    unsigned char sb = tex2D(solidTex2, xc, yc - 1);
    unsigned char st = tex2D(solidTex2, xc, yc + 1);
    if (sl)
      l = c;
    if (sr)
      r = c;
    if (sb)
      b = c;
    if (st)
      t = c;
    p[index] = (l + r + t + b + alpha * rhs) * 0.25;
  }
}

__global__ void __diffuseU(float *u, hermes::cuda::Grid2Info uInfo, float k,
                           float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * uInfo.resolution.x + x;
  if (x < uInfo.resolution.x && y < uInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float l = tex2D(uTex2, xc - 1, yc);
    float r = tex2D(uTex2, xc + 1, yc);
    float b = tex2D(uTex2, xc, yc - 1);
    float t = tex2D(uTex2, xc, yc + 1);
    float rhs = tex2D(uCopyTex2, xc, yc);
    unsigned char sl = tex2D(solidTex2, xc - 1, yc);
    unsigned char sr = tex2D(solidTex2, xc, yc);
    unsigned char sb = tex2D(solidTex2, xc, yc - 1);
    unsigned char st = tex2D(solidTex2, xc, yc + 1);
    if (sl)
      l = tex2D(uSolidTex2, xc - 1, yc);
    if (sr)
      r = tex2D(uSolidTex2, xc, yc);
    if (sb)
      b = tex2D(uSolidTex2, xc, yc - 1);
    if (st)
      t = tex2D(uSolidTex2, xc, yc + 1);
    float scale = dt * k / (uInfo.dx * uInfo.dx);
    u[index] = (scale * (l + r + t + b) + rhs) / (1 + 4 * scale);
  }
}

__global__ void __diffuseV(float *v, hermes::cuda::Grid2Info vInfo, float k,
                           float dt) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * vInfo.resolution.x + x;
  if (x < vInfo.resolution.x && y < vInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    float l = tex2D(vTex2, xc - 1, yc);
    float r = tex2D(vTex2, xc + 1, yc);
    float b = tex2D(vTex2, xc, yc - 1);
    float t = tex2D(vTex2, xc, yc + 1);
    float rhs = tex2D(vCopyTex2, xc, yc);
    unsigned char sl = tex2D(solidTex2, xc - 1, yc);
    unsigned char sr = tex2D(solidTex2, xc + 1, yc);
    unsigned char sb = tex2D(solidTex2, xc, yc - 1);
    unsigned char st = tex2D(solidTex2, xc, yc);
    if (sl)
      l = tex2D(uSolidTex2, xc - 1, yc);
    if (sr)
      r = tex2D(uSolidTex2, xc + 1, yc);
    if (sb)
      b = tex2D(uSolidTex2, xc, yc - 1);
    if (st)
      t = tex2D(uSolidTex2, xc, yc);
    float scale = dt * k / (vInfo.dx * vInfo.dx);
    v[index] = (scale * (l + r + t + b) + rhs) / (1 + 4 * scale);
  }
}

__global__ void __projectionStepU(float *u, hermes::cuda::Grid2Info uInfo,
                                  float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * uInfo.resolution.x + x;
  if (x < uInfo.resolution.x && y < uInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    if (tex2D(solidTex2, xc - 1, yc))
      u[index] = tex2D(uSolidTex2, xc - 1, yc);
    else if (tex2D(solidTex2, xc, yc))
      u[index] = tex2D(uSolidTex2, xc, yc);
    else {
      float l = tex2D(pressureTex2, xc - 1, yc);
      float r = tex2D(pressureTex2, xc, yc);
      u[index] -= scale * (r - l);
    }
  }
}

__global__ void __projectionStepV(float *v, hermes::cuda::Grid2Info vInfo,
                                  float scale) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * vInfo.resolution.x + x;
  if (x < vInfo.resolution.x && y < vInfo.resolution.y) {
    float xc = x + 0.5;
    float yc = y + 0.5;
    if (tex2D(solidTex2, xc, yc - 1))
      v[index] = tex2D(vSolidTex2, xc, yc - 1);
    else if (tex2D(solidTex2, xc, yc))
      v[index] = tex2D(vSolidTex2, xc, yc);
    else {
      float b = tex2D(pressureTex2, xc, yc - 1);
      float t = tex2D(pressureTex2, xc, yc);
      v[index] -= scale * (t - b);
    }
  }
}

void unbindTextures() {
  hipUnbindTexture(vTex2);
  hipUnbindTexture(uTex2);
  hipUnbindTexture(densityTex2);
  hipUnbindTexture(divergenceTex2);
  hipUnbindTexture(pressureTex2);
  hipUnbindTexture(solidTex2);
  hipUnbindTexture(uSolidTex2);
  hipUnbindTexture(vSolidTex2);
  hipUnbindTexture(uForceTex2);
  hipUnbindTexture(vForceTex2);
}

void bindTextures(const hermes::cuda::StaggeredGridTexture2 &velocity,
                  const hermes::cuda::StaggeredGridTexture2 &velocityCopy,
                  const hermes::cuda::GridTexture2<float> &density,
                  const hermes::cuda::GridTexture2<float> &divergence,
                  const hermes::cuda::GridTexture2<float> &pressure,
                  const hermes::cuda::GridTexture2<unsigned char> &solid,
                  const hermes::cuda::StaggeredGridTexture2 &forceField,
                  const hermes::cuda::StaggeredGridTexture2 &solidVelocity) {
  using namespace hermes::cuda;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(
      uCopyTex2, velocityCopy.u().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      vCopyTex2, velocityCopy.v().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      uTex2, velocity.u().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      vTex2, velocity.v().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      densityTex2, density.texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      divergenceTex2, divergence.texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      pressureTex2, pressure.texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      uSolidTex2, solidVelocity.u().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      vSolidTex2, solidVelocity.v().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      uForceTex2, forceField.u().texture().textureArray(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(
      vForceTex2, forceField.v().texture().textureArray(), channelDesc));
  channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex2, solid.texture().textureArray(),
                                    channelDesc));
}

void setupTextures() {
  // uTex2.addressMode[0] = hipAddressModeBorder;
  // uTex2.addressMode[1] = hipAddressModeBorder;
  uSolidTex2.filterMode = hipFilterModePoint;
  uSolidTex2.normalized = 0;
  vSolidTex2.filterMode = hipFilterModePoint;
  vSolidTex2.normalized = 0;
  uForceTex2.filterMode = hipFilterModePoint;
  uForceTex2.normalized = 0;
  vForceTex2.filterMode = hipFilterModePoint;
  vForceTex2.normalized = 0;
  uTex2.filterMode = hipFilterModeLinear;
  uTex2.normalized = 0;
  vTex2.filterMode = hipFilterModeLinear;
  vTex2.normalized = 0;
  uCopyTex2.filterMode = hipFilterModeLinear;
  uCopyTex2.normalized = 0;
  vCopyTex2.filterMode = hipFilterModeLinear;
  vCopyTex2.normalized = 0;
  densityTex2.filterMode = hipFilterModeLinear;
  densityTex2.normalized = 0;
  divergenceTex2.filterMode = hipFilterModePoint;
  divergenceTex2.normalized = 0;
  pressureTex2.filterMode = hipFilterModePoint;
  pressureTex2.normalized = 0;
  solidTex2.filterMode = hipFilterModePoint;
  solidTex2.normalized = 0;
}

void applyForceField(hermes::cuda::StaggeredGridTexture2 &velocity,
                     const hermes::cuda::StaggeredGridTexture2 &forceField,
                     float dt) {
  {
    auto info = velocity.v().info();
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __applyForceFieldV<<<td.gridSize, td.blockSize>>>(velocity.vDeviceData(),
                                                      velocity.v().info(), dt);
    velocity.v().texture().updateTextureMemory();
  }
  {
    auto info = velocity.u().info();
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __applyForceFieldU<<<td.gridSize, td.blockSize>>>(velocity.uDeviceData(),
                                                      velocity.u().info(), dt);
    velocity.u().texture().updateTextureMemory();
  }
}

void computeDivergence(const hermes::cuda::StaggeredGridTexture2 &velocity,
                       const hermes::cuda::GridTexture2<unsigned char> &solid,
                       hermes::cuda::GridTexture2<float> &divergence) {
  auto info = divergence.info();
  float invdx = 1.0 / info.dx;
  hermes::ThreadArrayDistributionInfo td(info.resolution);
  __computeDivergence<<<td.gridSize, td.blockSize>>>(
      divergence.texture().deviceData(), divergence.info(), invdx);
  divergence.texture().updateTextureMemory();
}

void computePressure(const hermes::cuda::GridTexture2<float> &divergence,
                     const hermes::cuda::GridTexture2<unsigned char> &solid,
                     hermes::cuda::GridTexture2<float> &pressure, float dt,
                     int iterations) {
  auto info = pressure.info();
  hermes::ThreadArrayDistributionInfo td(info.resolution);
  float alpha = -(info.dx * info.dx) / dt;
  for (int i = 0; i < iterations; i++) {
    pressure.texture().updateTextureMemory();
    __computePressure<<<td.gridSize, td.blockSize>>>(
        pressure.texture().deviceData(), pressure.info(), alpha);
    using namespace hermes::cuda;
    CUDA_CHECK(hipDeviceSynchronize());
  }
  pressure.texture().updateTextureMemory();
}

void diffuse(hermes::cuda::StaggeredGridTexture2 &velocity, float k, float dt,
             int iterations) {
  {
    auto info = velocity.u().info();
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    for (int i = 0; i < iterations; i++) {
      velocity.u().texture().updateTextureMemory();
      __diffuseU<<<td.gridSize, td.blockSize>>>(velocity.uDeviceData(), info, k,
                                                dt);
    }
  }
  {
    auto info = velocity.v().info();
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    for (int i = 0; i < iterations; i++) {
      velocity.v().texture().updateTextureMemory();
      __diffuseV<<<td.gridSize, td.blockSize>>>(velocity.vDeviceData(), info, k,
                                                dt);
    }
  }
  velocity.u().texture().updateTextureMemory();
  velocity.v().texture().updateTextureMemory();
}

void projectionStep(const hermes::cuda::GridTexture2<float> &pressure,
                    const hermes::cuda::GridTexture2<unsigned char> &solid,
                    hermes::cuda::StaggeredGridTexture2 &velocity, float dt) {
  {
    auto info = velocity.u().info();
    float invdx = 1.0 / info.dx;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepU<<<td.gridSize, td.blockSize>>>(
        velocity.uDeviceData(), velocity.u().info(), scale);
  }
  {
    auto info = velocity.v().info();
    float invdx = 1.0 / info.dx;
    float scale = dt * invdx;
    hermes::ThreadArrayDistributionInfo td(info.resolution);
    __projectionStepV<<<td.gridSize, td.blockSize>>>(
        velocity.vDeviceData(), velocity.v().info(), scale);
  }
  velocity.u().texture().updateTextureMemory();
  velocity.v().texture().updateTextureMemory();
}

} // namespace cuda

} // namespace poseidon
