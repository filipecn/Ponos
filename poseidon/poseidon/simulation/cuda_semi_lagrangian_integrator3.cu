#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 FilipeCN
 *
 * The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * iM the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

#include <hermes/storage/cuda_storage_utils.h>
#include <poseidon/simulation/cuda_integrator.h>

namespace poseidon {

namespace cuda {

texture<float, hipTextureType3D> uTex3;
texture<float, hipTextureType3D> vTex3;
texture<float, hipTextureType3D> wTex3;
texture<float, hipTextureType3D> phiTex3;
texture<unsigned char, hipTextureType3D> solidTex3;

__global__ void __advect(hermes::cuda::RegularGrid3Accessor<float> phi,
                         hermes::cuda::RegularGrid3Info uInfo,
                         hermes::cuda::RegularGrid3Info vInfo,
                         hermes::cuda::RegularGrid3Info wInfo, float dt) {
  using namespace hermes::cuda;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (phi.isIndexStored(x, y, z)) {
    unsigned char solid = tex3D(solidTex3, x + 0.5, y + 0.5, z + 0.5);
    if (solid) {
      phi(x, y, z) = 0;
      return;
    }
    point3f p = phi.worldPosition(x, y, z);
    point3f up = uInfo.toGrid(p) + vec3(0.5);
    point3f vp = vInfo.toGrid(p) + vec3(0.5);
    point3f wp = wInfo.toGrid(p) + vec3(0.5);
    vec3f vel(tex3D(uTex3, up.x, up.y, up.z), tex3D(vTex3, vp.x, vp.y, vp.z),
              tex3D(wTex3, vp.x, wp.y, wp.z));
    point3f pos = phi.gridPosition(p - vel * dt) + vec3(0.5);
    phi(x, y, z) = tex3D(phiTex3, pos.x, pos.y, pos.z);
  }
}

SemiLagrangianIntegrator3::SemiLagrangianIntegrator3() {
  uTex3.filterMode = hipFilterModeLinear;
  uTex3.normalized = 0;
  vTex3.filterMode = hipFilterModeLinear;
  vTex3.normalized = 0;
  wTex3.filterMode = hipFilterModeLinear;
  wTex3.normalized = 0;
  phiTex3.filterMode = hipFilterModeLinear;
  phiTex3.normalized = 0;
  solidTex3.filterMode = hipFilterModePoint;
  solidTex3.normalized = 0;
}

void SemiLagrangianIntegrator3::advect(
    hermes::cuda::VectorGrid3D &velocity,
    hermes::cuda::RegularGrid3<hermes::cuda::MemoryLocation::DEVICE,
                               unsigned char> &solid,
    hermes::cuda::RegularGrid3Df &phi, hermes::cuda::RegularGrid3Df &phiOut,
    float dt) {
  using namespace hermes::cuda;
  Array3<unsigned char> solidArray(solid.resolution());
  Array3<float> uArray(velocity.u().resolution());
  Array3<float> vArray(velocity.v().resolution());
  Array3<float> wArray(velocity.w().resolution());
  Array3<float> phiArray(phi.resolution());
  memcpy(solidArray, solid.data());
  memcpy(uArray, velocity.u().data());
  memcpy(vArray, velocity.v().data());
  memcpy(wArray, velocity.w().data());
  memcpy(phiArray, phi.data());
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
  CUDA_CHECK(hipBindTextureToArray(solidTex3, solidArray.data(), channelDesc));
  channelDesc = hipCreateChannelDesc<float>();
  CUDA_CHECK(hipBindTextureToArray(uTex3, uArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(vTex3, vArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(wTex3, wArray.data(), channelDesc));
  CUDA_CHECK(hipBindTextureToArray(phiTex3, phiArray.data(), channelDesc));
  hermes::ThreadArrayDistributionInfo td(phi.resolution());
  __advect<<<td.gridSize, td.blockSize>>>(
      phiOut.accessor(), velocity.u().info(), velocity.v().info(),
      velocity.w().info(), dt);
  hipUnbindTexture(solidTex3);
  hipUnbindTexture(vTex3);
  hipUnbindTexture(uTex3);
  hipUnbindTexture(wTex3);
  hipUnbindTexture(phiTex3);
}

} // namespace cuda

} // namespace poseidon