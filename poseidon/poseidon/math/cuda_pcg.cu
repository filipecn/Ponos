#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 FilipeCN
 *
 * The MIT License (MIT)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * iM the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

#include <hermes/numeric/cuda_blas.h>
#include <poseidon/math/cuda_pcg.h>

namespace poseidon {

namespace cuda {

using namespace hermes::cuda;

void pcg(MemoryBlock1Dd &x, FDMatrix2D &A, MemoryBlock1Dd &b,
         size_t maxNumberOfIterations, float tolerance) {
  // cpu memory
  MemoryBlock1Hd h_r(b.size(), 0);
  MemoryBlock1Hd h_z(b.size(), 0);
  MemoryBlock1Hd precon(b.size(), 0);
  // FDMatrix2H h_A(A.gridSize());
  // h_A.copy(A);
  // mic0(precon, h_A, 0.97, 0.25);
  // device memory
  MemoryBlock1Dd m;
  MemoryBlock1Dd r(b.size(), 0); // residual
  MemoryBlock1Dd z(b.size(), 0); // auxiliar
  MemoryBlock1Dd s(b.size(), 0); // search
  std::cerr << "max " << maxNumberOfIterations << std::endl;
  // r = b - A * x
  mul(A, x, r);
  sub(b, r, r);
  if (infnorm(r, m) <= tolerance)
    return;
  // z = M * r
  memcpy(z, r);
  // memcpy(h_r, r);
  // memcpy(h_z, z);
  // applyMIC0(h_A, precon, h_r, h_z);
  // memcpy(z, h_z);
  // s = z
  memcpy(s, z);
  // sigma = z '* r
  double sigma = dot(z, r, m);
  // std::cerr << "sigma = " << sigma << std::endl;
  size_t it = 0;
  // std::cerr << "S " << s << std::endl;
  while (it < maxNumberOfIterations) {
    // z = As
    mul(A, s, z);
    // std::cerr << "Z " << z << std::endl;
    // std::cerr << "S " << s << std::endl;
    // alpha = sigma / (z '* s)
    double alpha = sigma / dot(z, s, m);
    // std::cerr << "alpha " << alpha << std::endl;
    // x = alpha * s + x
    axpy(alpha, s, x, x);
    // r = r - alpha * z
    axpy(-alpha, z, r, r);
    // std::cerr << "r norm test\n";
    // std::cerr << r << std::endl;
    if (infnorm(r, m) <= tolerance) {
      std::cerr << "PCG RUN with " << it << " iterations.\n";
      return;
    }
    // z = M * r
    memcpy(z, r);
    // memcpy(h_r, r);
    // memcpy(h_z, z);
    // applyMIC0(h_A, precon, h_r, h_z);
    // memcpy(z, h_z);
    // sigmaNew = z '* r
    double sigmaNew = dot(z, r, m);
    // std::cerr << "sigmaNew " << sigmaNew << std::endl;
    // if (sigmaNew < tolerance * tolerance)
    //   break;
    // s = z + (sigmaNew / sigma) * s
    axpy(sigmaNew / sigma, s, z, s);
    sigma = sigmaNew;
    ++it;
  }
  // auto acc = h_A.accessor();
  // std::cerr << "BAD PCG!\n" << acc << std::endl;
  // std::cerr << b << std::endl;
  exit(-1);
}

void mic0(MemoryBlock1Hd &precon, FDMatrix3H &h_A, double tal, double sigma) {
#define SQR(A) ((A) * (A))
  auto A = h_A.accessor();
  auto E = precon.accessor();
  auto size = h_A.gridSize();
  auto indices = h_A.indexDataAccessor();
  for (size_t k = 0; k < size.z; k++)
    for (size_t j = 0; j < size.y; j++)
      for (size_t i = 0; i < size.x; i++)
        if (indices(i, j, k) >= 0) {
          double e =
              A(i, j, k, i, j, k) -
              ((indices.isIndexValid(i - 1, j, k) && indices(i - 1, j, k) >= 0)
                   ? SQR(A(i - 1, j, k, i, j, k) * E[indices(i - 1, j, k)])
                   : 0.0) -
              ((indices.isIndexValid(i, j - 1, k) && indices(i, j - 1, k) >= 0)
                   ? SQR(A(i, j - 1, k, i, j, k) * E[indices(i, j - 1, k)])
                   : 0.0) -
              ((indices.isIndexValid(i, j, k - 1) && indices(i, j, k - 1) >= 0)
                   ? SQR(A(i, j, k - 1, i, j, k) * E[indices(i, j, k - 1)])
                   : 0.0) -
              tal * (A(i - 1, j, k, i, j, k) *
                         (A(i - 1, j, k, i - 1, j + 1, k) +
                          A(i - 1, j, k, i - 1, j, k + 1)) *
                         ((indices.isIndexValid(i - 1, j, k) &&
                           indices(i - 1, j, k) >= 0)
                              ? SQR(E[indices(i - 1, j, k)])
                              : 0.0) +
                     A(i, j - 1, k, i, j, k) *
                         (A(i, j - 1, k, i + 1, j - 1, k) +
                          A(i, j - 1, k, i, j - 1, k + 1)) *
                         ((indices.isIndexValid(i, j - 1, k) &&
                           indices(i, j - 1, k) >= 0)
                              ? SQR(E[indices(i, j - 1, k)])
                              : 0.0) +
                     A(i, j, k - 1, i, j, k) *
                         (A(i, j, k - 1, i + 1, j, k - 1) +
                          A(i, j, k - 1, i, j + 1, k - 1)) *
                         ((indices.isIndexValid(i, j, k - 1) &&
                           indices(i, j, k - 1) >= 0)
                              ? SQR(E[indices(i, j, k - 1)])
                              : 0.0));
          if (e < sigma * A(i, j, k, i, j, k))
            e = A(i, j, k, i, j, k);
          E[indices(i, j, k)] = 1 / sqrt(e);
        }
}

void applyMIC0(FDMatrix3H &h_A, MemoryBlock1Hd &h_precon, MemoryBlock1Hd &h_r,
               MemoryBlock1Hd &h_z) {
  MemoryBlock1Hd h_q(h_z.size(), 0);
  auto indices = h_A.indexDataAccessor();
  auto A = h_A.accessor();
  auto precon = h_precon.accessor();
  auto r = h_r.accessor();
  auto z = h_z.accessor();
  auto q = h_q.accessor();
  auto size = h_A.gridSize();
  // solve Lq = r
  for (size_t k = 0; k < size.z; ++k)
    for (size_t j = 0; j < size.y; ++j)
      for (size_t i = 0; i < size.x; ++i)
        if (indices(i, j, k) >= 0) {
          double t =
              r[indices(i, j, k)] -
              ((indices.isIndexValid(i - 1, j, k) && indices(i - 1, j, k) >= 0)
                   ? A(i - 1, j, k, i, j, k) * precon[indices(i - 1, j, k)] *
                         q[indices(i - 1, j, k)]
                   : 0.0) -
              ((indices.isIndexValid(i, j - 1, k) && indices(i, j - 1, k) >= 0)
                   ? A(i, j - 1, k, i, j, k) * precon[indices(i, j - 1, k)] *
                         q[indices(i, j - 1, k)]
                   : 0.0) -
              ((indices.isIndexValid(i, j, k - 1) && indices(i, j, k - 1) >= 0)
                   ? A(i, j, k - 1, i, j, k) * precon[indices(i, j, k - 1)] *
                         q[indices(i, j, k - 1)]
                   : 0.0);
          q[indices(i, j, k)] = t * precon[indices(i, j, k)];
        }
  // solve L^Tz = q
  for (int k = (int)size.z - 1; k >= 0; --k)
    for (int j = (int)size.y - 1; j >= 0; --j)
      for (int i = (int)size.x - 1; i >= 0; --i)
        if (indices(i, j, k) >= 0) {
          double t =
              q[indices(i, j, k)] -
              ((indices.isIndexValid(i + 1, j, k) && indices(i + 1, j, k) >= 0)
                   ? A(i, j, k, i + 1, j, k) * precon[indices(i, j, k)] *
                         z[indices(i + 1, j, k)]
                   : 0.0) -
              ((indices.isIndexValid(i, j + 1, k) && indices(i, j + 1, k) >= 0)
                   ? A(i, j, k, i, j + 1, k) * precon[indices(i, j, k)] *
                         z[indices(i, j + 1, k)]
                   : 0.0) -
              ((indices.isIndexValid(i, j, k + 1) && indices(i, j, k + 1) >= 0)
                   ? A(i, j, k, i, j, k + 1) * precon[indices(i, j, k)] *
                         z[indices(i, j, k + 1)]
                   : 0.0);
          z[indices(i, j, k)] = t * precon[indices(i, j, k)];
        }
}

void pcg(MemoryBlock1Dd &x, FDMatrix3D &A, MemoryBlock1Dd &b,
         size_t maxNumberOfIterations, float tolerance) {
  // cpu memory
  MemoryBlock1Hd h_r(b.size(), 0);
  MemoryBlock1Hd h_z(b.size(), 0);
  MemoryBlock1Hd precon(b.size(), 0);
  FDMatrix3H h_A(A.gridSize());
  h_A.copy(A);
  mic0(precon, h_A, 0.97, 0.25);
  // device memory
  MemoryBlock1Dd m;
  MemoryBlock1Dd r(b.size(), 0); // residual
  MemoryBlock1Dd z(b.size(), 0); // auxiliar
  MemoryBlock1Dd s(b.size(), 0); // search
  std::cerr << "max " << maxNumberOfIterations << std::endl;
  // r = b - A * x
  mul(A, x, r);
  sub(b, r, r);
  if (infnorm(r, m) <= tolerance)
    return;
  // z = M * r
  memcpy(z, r);
  // memcpy(h_r, r);
  // memcpy(h_z, z);
  // applyMIC0(h_A, precon, h_r, h_z);
  // memcpy(z, h_z);
  // s = z
  memcpy(s, z);
  // sigma = z '* r
  double sigma = dot(z, r, m);
  // std::cerr << "sigma = " << sigma << std::endl;
  size_t it = 0;
  // std::cerr << "S " << s << std::endl;
  while (it < maxNumberOfIterations) {
    // z = As
    mul(A, s, z);
    // std::cerr << "Z " << z << std::endl;
    // std::cerr << "S " << s << std::endl;
    // alpha = sigma / (z '* s)
    double alpha = sigma / dot(z, s, m);
    // std::cerr << "alpha " << alpha << std::endl;
    // x = alpha * s + x
    axpy(alpha, s, x, x);
    // r = r - alpha * z
    axpy(-alpha, z, r, r);
    // std::cerr << "r norm test\n";
    // std::cerr << r << std::endl;
    if (infnorm(r, m) <= tolerance) {
      std::cerr << "PCG RUN with " << it << " iterations.\n";
      return;
    }
    // z = M * r
    memcpy(z, r);
    // memcpy(h_r, r);
    // memcpy(h_z, z);
    // applyMIC0(h_A, precon, h_r, h_z);
    // memcpy(z, h_z);
    // sigmaNew = z '* r
    double sigmaNew = dot(z, r, m);
    // std::cerr << "sigmaNew " << sigmaNew << std::endl;
    // if (sigmaNew < tolerance * tolerance)
    //   break;
    // s = z + (sigmaNew / sigma) * s
    axpy(sigmaNew / sigma, s, z, s);
    sigma = sigmaNew;
    ++it;
  }
  auto acc = h_A.accessor();
  std::cerr << "BAD PCG!\n" << acc << std::endl;
  std::cerr << b << std::endl;
  exit(-1);
}

} // namespace cuda

} // namespace poseidon
